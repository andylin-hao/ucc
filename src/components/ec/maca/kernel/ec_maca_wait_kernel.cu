#include "hip/hip_runtime.h"
#ifndef UINT32_MAX
#define __STDC_LIMIT_MACROS
#include <stdint.h>
#endif

#ifdef __cplusplus
extern "C" {
#endif
#include "../ec_maca.h"
#ifdef __cplusplus
}
#endif

__global__ void wait_kernel(volatile ucc_ec_maca_executor_state_t *state) {
    ucc_ec_maca_executor_state_t st;

    *state = UCC_EC_MACA_EXECUTOR_STARTED;
    do {
        st = *state;
    } while (st != UCC_EC_MACA_EXECUTOR_SHUTDOWN);
    *state = UCC_EC_MACA_EXECUTOR_SHUTDOWN_ACK;
    return;
}

#ifdef __cplusplus
extern "C" {
#endif

ucc_status_t
ucc_ec_maca_post_kernel_stream_task(ucc_ec_maca_executor_state_t *state,
                                    mcStream_t stream)
{
    wait_kernel<<<1, 1, 0, stream>>>(state);
    MACA_CHECK(mcGetLastError());
    return UCC_OK;
}

#ifdef __cplusplus
}
#endif